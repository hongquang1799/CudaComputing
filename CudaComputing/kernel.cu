#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include "iostream"
#include "time.h"

#include <chrono>
#include <stdio.h>

#include <assert.h>

float cpu_sigmoid(float x)
{
	return 1.0f / (1.0f + expf(x));
}

void CPU_AddVector(float * a, float * b, float * c, float n)
{
	int i;

	for (i = 0; i < n; i++)
	{
		c[i] = (a[i] + b[i]);
	}
}

__device__ float gpu_sigmoid(float x)
{
	return 1.0f / (1.0f + expf(x));
}

__global__ void GPU_AddVector(float * a, float * b, float * c, int n)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;

	if (i < n)
		c[i] = (a[i] + b[i]);
}

void main()
{
	srand(time(NULL));

	size_t n = 1 << 10;
	size_t n_bytes = sizeof(float) * n;

	float * a = (float *)malloc(n_bytes);
	float * b = (float *)malloc(n_bytes);
	float * c = (float *)malloc(n_bytes);

	float *d_a, *d_b, *d_c;
	
	hipMalloc(&d_a, n_bytes);
	hipMalloc(&d_b, n_bytes);
	hipMalloc(&d_c, n_bytes);

	for (int i = 0; i < n; i++)
	{
		a[i] = rand() % 10;
		b[i] = rand() % 10;
	}

	hipMemcpy(d_a, a, n_bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, n_bytes, hipMemcpyHostToDevice);

	size_t n_iter = 10000;

	// cpu computing
	auto start = std::chrono::high_resolution_clock::now();
	
	for (int i = 0; i < n_iter; i++)
	{
		CPU_AddVector(a, b, c, n);
	}
		
	
	auto stop = std::chrono::high_resolution_clock::now();
	auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(stop - start);
	std::cout << "Speed of CPU: " << duration.count() << " micro seconds" << std::endl;


	// gpu computing
	GPU_AddVector << <1, 1024>> >(d_a, d_b, d_c, n);
	start = std::chrono::high_resolution_clock::now();

	for (int i = 0; i < n_iter; i++)
	{
		GPU_AddVector<<<1, 1024>>>(d_a, d_b, d_c, n);
	}
	hipDeviceSynchronize();

	stop = std::chrono::high_resolution_clock::now();
	duration = std::chrono::duration_cast<std::chrono::milliseconds>(stop - start);
	std::cout << "Speed of GPU: " << duration.count() << " micro seconds" << std::endl;


	// Warmup
	//hipEvent_t launch_begin, launch_end;
	//hipEventCreate(&launch_begin);
	//hipEventCreate(&launch_end);
	//GPU_AddVector<<<1, n>>>(d_a, d_b, d_c, n);// num blocks, num_threads
	//float total_time = 0;
	//int num_times = 100;
	//// Get average of 100 runs
	//for (int i = 0; i<num_times; i++) {
	//	hipEventRecord(launch_begin, 0);
	//	GPU_AddVector <<<1, n >>>(d_a, d_b, d_c, n);
	//	hipEventRecord(launch_end, 0);
	//	hipEventSynchronize(launch_end);
	//	float time = 0;
	//	hipEventElapsedTime(&time, launch_begin, launch_end);
	//	total_time += time;
	//}
	//std::cout << "Speed of CPU vector Addition: " << total_time << " micro seconds" << std::endl;

	hipMemcpy(c, d_c, n_bytes, hipMemcpyDeviceToHost);

	for (int i = 0; i < 5; i++)
	{
		printf("%f + %f = %f\n", a[i], b[i], c[i]);
	}



	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	free(a);
	free(b);
	free(c);

	getchar();
}